#include "hip/hip_runtime.h"
#include "config.h"
#include "preprocess.h"

static uint8_t *img_buffer_device = nullptr;


void cuda_preprocess_init(int max_image_size)
{
    // 申请设备内存
    CUDA_CHECK(hipMalloc((void**)&img_buffer_device, max_image_size * 3));
}

void cuda_preprocess_destroy()
{
    CUDA_CHECK(hipFree(img_buffer_device));
}

void cuda_preprocess(uint8_t *src, int src_width, int src_height, float *dst, int dst_width, int dst_height)
{
    int img_size = src_width * src_height * 3;
    CUDA_CHECK(hipMemcpy(img_buffer_device, src, img_size, hipMemcpyHostToDevice))
}

void process_input(cv::Mat &src, float *input_device_buffer)
{
    cuda_preprocess(src.ptr(), src.cols, src.rows, input_device_buffer, kInputW, kInputH);
}